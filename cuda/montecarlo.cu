
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <assert.h>

//#include <cutil_inline.h>

//#include <culapack.h>
//#include <culapackdevice.h>

#define imin(X, Y)  ((X) < (Y) ? (X) : (Y))

__device__ inline float MoroInvCNDgpu(float P){
    const float a1 = 2.50662823884f;
    const float a2 = -18.61500062529f;
    const float a3 = 41.39119773534f;
    const float a4 = -25.44106049637f;
    const float b1 = -8.4735109309f;
    const float b2 = 23.08336743743f;
    const float b3 = -21.06224101826f;
    const float b4 = 3.13082909833f;
    const float c1 = 0.337475482272615f;
    const float c2 = 0.976169019091719f;
    const float c3 = 0.160797971491821f;
    const float c4 = 2.76438810333863E-02f;
    const float c5 = 3.8405729373609E-03f;
    const float c6 = 3.951896511919E-04f;
    const float c7 = 3.21767881768E-05f;
    const float c8 = 2.888167364E-07f;
    const float c9 = 3.960315187E-07f;
    float y, z;

    if(P <= 0 || P >= 1.0f)
        return __int_as_float(0x7FFFFFFF);

    y = P - 0.5f;
    if(fabsf(y) < 0.42f){
        z = y * y;
        z = y * (((a4 * z + a3) * z + a2) * z + a1) / ((((b4 * z + b3) * z + b2) * z + b1) * z + 1.0f);
    }else{
        if(y > 0)
            z = __logf(-__logf(1.0f - P));
        else
            z = __logf(-__logf(P));

        z = c1 + z * (c2 + z * (c3 + z * (c4 + z * (c5 + z * (c6 + z * (c7 + z * (c8 + z * c9)))))));
        if(y < 0) z = -z;
    }

    return z;
}

__global__ void NormalDistribution(float *A, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        A[i] = MoroInvCNDgpu(A[i]);
}

__global__ void CumSum(float *A, int N)
{
    float *B = A + (threadIdx.x)*N;
    float cumsum = 0;
    for(int i=0; i<N; i++){
        cumsum += B[i];
        B[i] = cumsum;
    }  
}

void UniformDistribution(float *A, int M, int N)
{
	int i,j;
	for(i=0; i<M; i++){
		for(j=0; j<N; j++){
			A[i+j*M] = (float) rand() / RAND_MAX;
		}
	}
}


void OutputMatrix(char *fileName, float *A, int M, int N){
    FILE *fp;
    fp = fopen(fileName, "w");

    int i,j;
    for(i=0; i<M; i++){
        for(j=0; j<N; j++){
            fprintf(fp, "%f ", A[i+j*M]);
        }
        fprintf(fp, "\n");
    } 

    fclose(fp);
}

int main(){

    srand(time(0));

    int n = 1000;
    int N = 16;

    int length = n*(N-1);
    size_t size = length*sizeof(float);

    float *dW = (float*) malloc( size );
    UniformDistribution(dW, n, N-1);

    float *dW_d;
    hipMalloc( (void**) &dW_d, size);
    hipMemcpy(dW_d, dW, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (n*N + threadsPerBlock - 1) / threadsPerBlock;
    
    NormalDistribution<<<blocksPerGrid, threadsPerBlock>>>(dW_d, length); 
    
    hipMemcpy(dW, dW_d, size, hipMemcpyDeviceToHost);
    hipFree(dW_d);

    OutputMatrix("dW_gpu.mat", dW, n, N-1); 

    free(dW);

    dW = (float*) malloc(size);
    
    for(int i=0; i<N-1; i++){
        for(int j=0; j<n; j++){
            dW[i*n+j] = j;
        }
    }

    OutputMatrix("init_scan.mat", dW, n, N-1);

    hipMalloc( (void**) &dW_d, size);
    hipMemcpy(dW_d, dW, size, hipMemcpyHostToDevice);
    
    CumSum<<<1, N-1>>>(dW_d, n);     
    
    hipMemcpy(dW, dW_d, size, hipMemcpyDeviceToHost);
    hipFree(dW_d);

    OutputMatrix("scan_gpu.mat", dW, n, N-1); 

    free(dW); 

     

}

